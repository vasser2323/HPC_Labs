#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <vector>
#include <random>
#include <chrono>

// Function for CPU matrix multiplication
void matrix_mul_CPU(const std::vector<double>& A, const std::vector<double>& B, std::vector<double>& C, int rowsA, int colsA, int colsB) {
    for (int i = 0; i < rowsA; i++) {
        for (int j = 0; j < colsB; j++) {
            double sum = 0.0;
            for (int k = 0; k < colsA; k++)
                sum += A[i * colsA + k] * B[k * colsB + j];
            C[i * colsB + j] = sum;
        }
    }
}

// Kernel for GPU matrix multiplication
__global__ void matrix_mul_GPU(const double* A, const double* B, double* C, int rowsA, int colsA, int colsB) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    double sum = 0.0;

    if (row < rowsA && col < colsB) {
        for (int k = 0; k < colsA; k++) {
            sum += A[row * colsA + k] * B[k * colsB + col];
        }
        C[row * colsB + col] = sum;
    }
}

int main() {
    std::default_random_engine generator(std::chrono::system_clock::now().time_since_epoch().count());
    std::uniform_real_distribution<double> distribution(0.0, 1.0);

    int m = 1000;
    int n = 1000;
    int p = 1000;

    std::vector<double> a(m * p);
    std::vector<double> b(p * n);
    std::vector<double> c(m * n, 0.0);

    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < p; ++j) {
            a[i * p + j] = distribution(generator);
        }
    }

    for (int i = 0; i < p; ++i) {
        for (int j = 0; j < n; ++j) {
            b[i * n + j] = distribution(generator);
        }
    }

    std::vector<double> hc(m * n, 0.0);

    // CPU matrix multiplication
    auto start_cpu = std::chrono::high_resolution_clock::now();
    matrix_mul_CPU(a, b, c, m, n, p);
    auto end_cpu = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> cpu_duration = end_cpu - start_cpu;

    double* da, * db, * dc;
    hipMalloc(&da, m * p * sizeof(double));
    hipMalloc(&db, p * n * sizeof(double));
    hipMalloc(&dc, m * n * sizeof(double));

    hipMemcpy(da, a.data(), m * p * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(db, b.data(), p * n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dc, hc.data(), m * n * sizeof(double), hipMemcpyHostToDevice);

    dim3 block_dim(32, 32);
    dim3 grid_dim((n + block_dim.x - 1) / block_dim.x, (m + block_dim.y - 1) / block_dim.y);

    hipEvent_t begin, stop;
    hipEventCreate(&begin);
    hipEventCreate(&stop);

    hipEventRecord(begin, 0);
    matrix_mul_GPU << <grid_dim, block_dim >> > (da, db, dc, m, n, p);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float gpu_time;
    hipEventElapsedTime(&gpu_time, begin, stop);

    hipMemcpy(hc.data(), dc, m * n * sizeof(double), hipMemcpyDeviceToHost);

    // Verify results
    bool is_equal = true;
    for (int i = 0; i < m * n; ++i) {
        if (c[i] != hc[i]) {
            is_equal = false;
            break;
        }
    }

    std::cout << "CPU time (s): " << cpu_duration.count() << std::endl;
    std::cout << "GPU time (s): " << gpu_time / 1000.0 << std::endl;

    hipFree(da);
    hipFree(db);
    hipFree(dc);

    hipEventDestroy(begin);
    hipEventDestroy(stop);

    return 0;
}
